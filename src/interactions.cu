#include "hip/hip_runtime.h"
#include "interactions.h"
#include "utilities.h"

#include <thrust/random.h>

__host__ __device__ glm::vec3 calculate_random_direction_in_hemisphere(
    glm::vec3 normal,
    thrust::default_random_engine& rng) {
  thrust::uniform_real_distribution<float> u01(0, 1);

  float up = sqrt(u01(rng));       // cos(theta)
  float over = sqrt(1 - up * up);  // sin(theta)
  float around = u01(rng) * TWO_PI;

  // Find a direction that is not the normal based off of whether or not the normal's components are
  // all equal to sqrt(1/3) or whether or not at least one component is less than sqrt(1/3). Learned
  // this trick from Peter Kutz.
  glm::vec3 direction_not_normal;
  if (abs(normal.x) < SQRT_OF_ONE_THIRD) {
    direction_not_normal = glm::vec3(1, 0, 0);
  } else if (abs(normal.y) < SQRT_OF_ONE_THIRD) {
    direction_not_normal = glm::vec3(0, 1, 0);
  } else {
    direction_not_normal = glm::vec3(0, 0, 1);
  }

  // Use not-normal direction to generate two perpendicular directions
  glm::vec3 perpendicular_direction_1 = glm::normalize(glm::cross(normal, direction_not_normal));
  glm::vec3 perpendicular_direction_2 =
      glm::normalize(glm::cross(normal, perpendicular_direction_1));

  return up * normal + cos(around) * over * perpendicular_direction_1 +
         sin(around) * over * perpendicular_direction_2;
}

__host__ __device__ void scatter_ray(PathSegment& path_segment,
                                     glm::vec3 intersection_point,
                                     glm::vec3 normal,
                                     const Material& mat,
                                     thrust::default_random_engine& rng) {
  Ray& ray = path_segment.ray;

  // Offset a little from the intersection surface
  ray.direction = calculate_random_direction_in_hemisphere(normal, rng);
  ray.origin = intersection_point + EPSILON * ray.direction;

  // TODO(aczw): figure out color contribution calculation
  path_segment.color *= mat.color;

  path_segment.remaining_bounces -= 1;
}
