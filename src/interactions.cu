#include "hip/hip_runtime.h"
#include "interactions.h"
#include "utilities.cuh"

#include <thrust/random.h>

#include <numbers>

#define SQRT_ONE_THIRD 0.5773502691896257645091487805019574556476f

__host__ __device__ glm::vec3 calculate_random_direction_in_hemisphere(
    glm::vec3 normal,
    thrust::default_random_engine& rng) {
  thrust::uniform_real_distribution<float> u01(0, 1);

  float up = sqrt(u01(rng));       // cos(theta)
  float over = sqrt(1 - up * up);  // sin(theta)
  float around = u01(rng) * 2.f * std::numbers::pi;

  // Find a direction that is not the normal based off of whether or not the normal's components are
  // all equal to sqrt(1/3) or whether or not at least one component is less than sqrt(1/3). Learned
  // this trick from Peter Kutz.
  glm::vec3 direction_not_normal;
  if (abs(normal.x) < SQRT_ONE_THIRD) {
    direction_not_normal = glm::vec3(1, 0, 0);
  } else if (abs(normal.y) < SQRT_ONE_THIRD) {
    direction_not_normal = glm::vec3(0, 1, 0);
  } else {
    direction_not_normal = glm::vec3(0, 0, 1);
  }

  // Use not-normal direction to generate two perpendicular directions
  glm::vec3 perp_dir_1 = glm::normalize(glm::cross(normal, direction_not_normal));
  glm::vec3 perp_dir_2 = glm::normalize(glm::cross(normal, perp_dir_1));

  return up * normal + cos(around) * over * perp_dir_1 + sin(around) * over * perp_dir_2;
}

__host__ __device__ void scatter_ray(PathSegment& path_segment,
                                     glm::vec3 intersection_point,
                                     glm::vec3 normal,
                                     const Material& mat,
                                     thrust::default_random_engine& rng) {
  Ray& ray = path_segment.ray;

  // Offset a little from the intersection surface
  ray.direction = calculate_random_direction_in_hemisphere(normal, rng);
  ray.origin = intersection_point + EPSILON * ray.direction;

  // TODO(aczw): figure out color contribution calculation
  path_segment.throughput *= mat.color;

  path_segment.remaining_bounces -= 1;
}
