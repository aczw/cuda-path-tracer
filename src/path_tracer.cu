#include "hip/hip_runtime.h"
#include "interactions.h"
#include "intersections.h"
#include "path_tracer.h"
#include "scene.h"
#include "scene_structs.h"
#include "utilities.cuh"

#include <hip/hip_runtime.h>
#include <cuda/std/limits>
#include <cuda/std/optional>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include <glm/glm.hpp>
#include <glm/gtx/norm.hpp>

#include <cmath>
#include <cstdio>
#include <numbers>

#define ERRORCHECK 1
#define FILENAME \
  (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define check_cuda_error(msg) check_cuda_error_function(msg, FILENAME, __LINE__)

void check_cuda_error_function(const char* msg, const char* file, int line) {
#if ERRORCHECK
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (hipSuccess == err) {
    return;
  }

  fprintf(stderr, "CUDA error");
  if (file) {
    fprintf(stderr, " (%s:%d)", file, line);
  }
  fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
  getchar();
#endif  // _WIN32
  exit(EXIT_FAILURE);
#endif  // ERRORCHECK
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void send_image_to_pbo(uchar4* pbo,
                                  glm::ivec2 resolution,
                                  int iter,
                                  glm::vec3* image) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int index = x + (y * resolution.x);
    glm::vec3 pix = image[index];

    glm::ivec3 color;
    color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
    color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
    color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

    // Each thread writes one pixel location in the texture (textel)
    pbo[index].w = 0;
    pbo[index].x = color.x;
    pbo[index].y = color.y;
    pbo[index].z = color.z;
  }
}

// TODO(aczw): convert to thrust::device_ptr? would need to use
// thrust::raw_pointer_cast when submitting these to kernels
static Scene* hst_scene = nullptr;
static GuiDataContainer* gui_data = nullptr;
static glm::vec3* dev_image = nullptr;
static Geometry* dev_geometry = nullptr;
static Material* dev_materials = nullptr;
static PathSegment* dev_path_segments = nullptr;
static cuda::std::optional<ShadingData>* dev_shading_data = nullptr;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void init_data_container(GuiDataContainer* imgui_data) {
  gui_data = imgui_data;
}

/**
 * Generate `PathSegment`s with rays from the camera through the screen into the
 * scene, which is the first bounce of rays.
 *
 * Antialiasing - add rays for sub-pixel sampling
 * motion blur - jitter rays "in time"
 * lens effect - jitter ray origin positions based on a lens
 */
__global__ void generate_ray_from_camera(Camera cam,
                                         int iter,
                                         int trace_depth,
                                         PathSegment* path_segments) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < cam.resolution.x && y < cam.resolution.y) {
    int index = x + (y * cam.resolution.x);
    PathSegment& segment = path_segments[index];

    segment.ray.origin = cam.position;
    segment.radiance = glm::vec3();
    segment.throughput = glm::vec3(1.0f);

    // TODO: implement antialiasing by jittering the ray
    segment.ray.direction =
        glm::normalize(cam.view -
                       cam.right * cam.pixelLength.x *
                           ((float)x - (float)cam.resolution.x * 0.5f) -
                       cam.up * cam.pixelLength.y *
                           ((float)y - (float)cam.resolution.y * 0.5f));

    segment.pixel_index = index;
    segment.remaining_bounces = trace_depth;
  }
}

/**
 * Generates shading data only. Generating new rays from this data is handled in
 * the shaders.
 */
__global__ void compute_intersections(
    int depth,
    int num_paths,
    PathSegment* path_segments,
    Geometry* geometry,
    int geometry_size,
    cuda::std::optional<ShadingData>* shading_data) {
  int path_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (path_index >= num_paths) {
    return;
  }

  Ray path_ray = path_segments[path_index].ray;

  float t_min = cuda::std::numeric_limits<float>::max();
  int hit_geometry_index = -1;
  glm::vec3 surface_normal;

  // TODO(aczw): do something with this value
  bool is_outside = true;

  // Naively parse through global geometry
  // TODO(aczw): use better intersection algorithm i.e. acceleration structures
  for (int geometry_index = 0; geometry_index < geometry_size;
       ++geometry_index) {
    Geometry& geom = geometry[geometry_index];
    cuda::std::optional<Intersection> curr_intersection_opt;

    switch (geom.type) {
      case Geometry::Type::Cube:
        curr_intersection_opt = cube_intersection_test(geom, path_ray);
        break;

      case Geometry::Type::Sphere:
        curr_intersection_opt = sphere_intersection_test(geom, path_ray);
        break;

      default:
        break;
    }

    // Compute the minimum t to determine what scene geometry object is the
    // closest
    if (curr_intersection_opt && t_min > curr_intersection_opt->t) {
      const Intersection& intersection = curr_intersection_opt.value();

      t_min = intersection.t;
      hit_geometry_index = geometry_index;
      surface_normal = intersection.surface_normal;
    }
  }

  if (hit_geometry_index == -1) {
    // Intersection calculation went out of bounds, path ends here
    shading_data[path_index] = cuda::std::nullopt;
  } else {
    ShadingData data;
    data.t = t_min;
    data.material_id = geometry[hit_geometry_index].material_id;
    data.surface_normal = surface_normal;

    shading_data[path_index] = data;
  }
}

/**
 * "Fake" shader demonstrating what you might do with the info in a
 * `ShadingData`, as well as how to use Thrust's random number generator.
 * Observe that since the Thrust random number generator basically adds "noise"
 * to the iteration, the image should start off noisy and get cleaner as more
 * iterations are computed.
 *
 * Note that this shader does NOT do a BSDF evaluation! Your shaders should
 * handle that - this can allow techniques such as bump mapping.
 */
__global__ void shade_fake_material(
    int curr_iteration,
    int num_paths,
    cuda::std::optional<ShadingData>* shading_data,
    PathSegment* path_segments,
    Material* materials) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= num_paths) {
    return;
  }

  cuda::std::optional<ShadingData> data_opt = shading_data[index];

  // If there was no intersection, color the ray black. Lots of renderers use 4
  // channel color, RGBA, where A = alpha, often used for opacity, in which case
  // they can indicate no opacity. This can be useful for post-processing and
  // image compositing.
  if (!data_opt) {
    path_segments[index].throughput = glm::vec3();
    return;
  }

  const ShadingData& data = data_opt.value();

  // Set up the RNG. LOOK: this is how you use thrust's RNG! Please look at
  // make_seeded_random_engine as well.
  thrust::default_random_engine rng =
      make_seeded_random_engine(curr_iteration, index, 0);
  thrust::uniform_real_distribution<float> u01(0, 1);

  Material material = materials[data.material_id];
  glm::vec3 material_color = material.color;

  // If the material indicates that the object was a light, "light" the ray
  if (material.emittance > 0.f) {
    path_segments[index].throughput *= material_color * material.emittance;
  } else {
    // Otherwise, do some pseudo-lighting computation. This is actually more
    // like what you would expect from shading in a rasterizer like OpenGL.
    // TODO: replace this! you should be able to start with basically a
    // one-liner
    float lightTerm =
        glm::dot(data.surface_normal, glm::vec3(0.0f, 1.0f, 0.0f));
    path_segments[index].throughput *=
        (material_color * lightTerm) * 0.3f +
        ((1.0f - data.t * 0.02f) * material_color) * 0.7f;
    path_segments[index].throughput *=
        u01(rng);  // apply some noise because why not
  }
}

__global__ void shade_material(int curr_iteration,
                               int num_paths,
                               int curr_depth,
                               cuda::std::optional<ShadingData>* shading_data,
                               PathSegment* path_segments,
                               Material* materials) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= num_paths) {
    return;
  }

  cuda::std::optional<ShadingData> data_opt = shading_data[index];

  if (!data_opt) {
    return;
  }

  const ShadingData& data = data_opt.value();

  // Set up the RNG. LOOK: this is how you use thrust's RNG! Please look at
  // make_seeded_random_engine as well.
  thrust::default_random_engine rng =
      make_seeded_random_engine(curr_iteration, index, curr_depth);
  thrust::uniform_real_distribution<float> u01(0, 1);

  Material material = materials[data.material_id];
  glm::vec3 material_color = material.color;

  if (material.emittance > 0.f) {
    // If the material indicates that the object was a light, "light" the ray.
    // This also indicates that this path is complete
    path_segments[index].radiance =
        material.emittance * path_segments[index].throughput;
  } else {
    glm::vec3 omega_o = -path_segments[index].ray.direction;

    // Calculate simple Lambertian lighting
    glm::vec3 bsdf = material_color * static_cast<float>(std::numbers::inv_pi);

    // Cosine-weighted hemisphere sampling
    float pdf =
        glm::abs(glm::dot(data.surface_normal, omega_o) /
                 (glm::length(data.surface_normal) * glm::length(omega_o))) /
        std::numbers::pi;

    float lambert = glm::abs(glm::dot(data.surface_normal, omega_o));

    path_segments[index].throughput *= bsdf * lambert / pdf;
  }

  const Ray& original_ray = path_segments[index].ray;

  // Determine next ray
  glm::vec3 new_direction =
      calculate_random_direction_in_hemisphere(data.surface_normal, rng);
  path_segments[index].ray.origin = original_ray.origin +
                                    (data.t * original_ray.direction) +
                                    (EPSILON * data.surface_normal);
  path_segments[index].ray.direction = new_direction;
}

/**
 * Add the current iteration's output to the overall image
 */
__global__ void final_gather(int num_paths,
                             glm::vec3* image,
                             PathSegment* path_segments) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index >= num_paths) {
    return;
  }

  const PathSegment& segment = path_segments[index];
  image[segment.pixel_index] += segment.radiance;
}

namespace path_tracer {

void initialize(Scene* scene) {
  hst_scene = scene;

  const Camera& cam = hst_scene->state.camera;
  const int pixel_count = cam.resolution.x * cam.resolution.y;

  hipMalloc(&dev_image, pixel_count * sizeof(glm::vec3));
  hipMemset(dev_image, 0, pixel_count * sizeof(glm::vec3));

  hipMalloc(&dev_path_segments, pixel_count * sizeof(PathSegment));

  hipMalloc(&dev_geometry, scene->geoms.size() * sizeof(Geometry));
  hipMemcpy(dev_geometry, scene->geoms.data(),
             scene->geoms.size() * sizeof(Geometry), hipMemcpyHostToDevice);

  hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  hipMemcpy(dev_materials, scene->materials.data(),
             scene->materials.size() * sizeof(Material),
             hipMemcpyHostToDevice);

  hipMalloc(&dev_shading_data,
             pixel_count * sizeof(cuda::std::optional<ShadingData>));
  hipMemset(dev_shading_data, 0,
             pixel_count * sizeof(cuda::std::optional<ShadingData>));

  check_cuda_error("path_trace_init");
}

void free() {
  // No-op if dev_image is null
  hipFree(dev_image);
  hipFree(dev_path_segments);
  hipFree(dev_geometry);
  hipFree(dev_materials);
  hipFree(dev_shading_data);

  check_cuda_error("path_trace_free");
}

void run(uchar4* pbo, int curr_iteration) {
  const int trace_depth = hst_scene->state.trace_depth;
  const Camera& camera = hst_scene->state.camera;
  const int num_pixels = camera.resolution.x * camera.resolution.y;

  // 2D block for generating ray from camera
  const dim3 block_size_2d(8, 8);
  const dim3 blocks_per_grid_2d(
      (camera.resolution.x + block_size_2d.x - 1) / block_size_2d.x,
      (camera.resolution.y + block_size_2d.y - 1) / block_size_2d.y);

  // 1D block for path tracing
  const int block_size_1d = 128;

  // Initialize `dev_path_segments` by using rays that come out of the camera.
  generate_ray_from_camera<<<blocks_per_grid_2d, block_size_2d>>>(
      camera, curr_iteration, trace_depth, dev_path_segments);
  check_cuda_error("generate_ray_from_camera");

  int curr_depth = 0;
  PathSegment* dev_path_segments_end = dev_path_segments + num_pixels;
  int num_paths = dev_path_segments_end - dev_path_segments;

  // Shoot ray into scene, bounce between objects, push shading chunks
  while (true) {
    // Clean shading chunks
    hipMemset(dev_shading_data, 0,
               num_pixels * sizeof(cuda::std::optional<ShadingData>));

    // Tracing
    dim3 num_blocks_path_segment_tracing =
        (num_paths + block_size_1d - 1) / block_size_1d;
    compute_intersections<<<num_blocks_path_segment_tracing, block_size_1d>>>(
        curr_depth, num_paths, dev_path_segments, dev_geometry,
        hst_scene->geoms.size(), dev_shading_data);
    check_cuda_error("compute_intersections: trace one bounce");
    hipDeviceSynchronize();
    curr_depth++;

    // TODO(aczw): stream compaction away dead paths here (for now, this means
    // `shading_data` contains `cuda::std::nullopt` instead of actual data)
    //
    // Note that you can't really use a 2D kernel launch any more - switch to
    // 1D.

    // TODO:
    // --- Shading Stage ---
    // Shade path segments based on shading_data and generate new rays by
    // evaluating the BSDF.
    // Start off with just a big kernel that handles all the different
    // materials you have in the scenefile.
    // TODO: compare between directly shading the path segments and shading
    // path segments that have been reshuffled to be contiguous in memory.
    shade_material<<<num_blocks_path_segment_tracing, block_size_1d>>>(
        curr_iteration, num_paths, curr_depth, dev_shading_data,
        dev_path_segments, dev_materials);

    // TODO(aczw): should be based off of stream compaction results (i.e. all
    // paths have been stream compacted away)
    if (curr_depth > 7) {
      break;
    }

    if (gui_data) {
      gui_data->traced_depth = curr_depth;
    }
  }

  // Assemble this iteration and apply it to the image
  dim3 num_blocks_pixels = (num_pixels + block_size_1d - 1) / block_size_1d;
  final_gather<<<num_blocks_pixels, block_size_1d>>>(num_paths, dev_image,
                                                     dev_path_segments);

  // Send results to OpenGL buffer for rendering
  send_image_to_pbo<<<blocks_per_grid_2d, block_size_2d>>>(
      pbo, camera.resolution, curr_iteration, dev_image);

  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image,
             num_pixels * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  check_cuda_error("path_trace");
}

}  // namespace path_tracer
