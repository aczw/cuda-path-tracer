#include "hip/hip_runtime.h"
#include "sample.cuh"
#include "utilities.cuh"

#include <numbers>

#define SQRT_ONE_THIRD 0.5773502691896257645091487805019574556476f

__host__ __device__ glm::vec3 calculate_random_direction_in_hemisphere(
    glm::vec3 normal,
    thrust::default_random_engine& rng) {
  thrust::uniform_real_distribution<float> uniform_01;

  float up = std::sqrt(uniform_01(rng));  // cos(theta)
  float over = std::sqrt(1 - up * up);    // sin(theta)
  float around = uniform_01(rng) * 2.f * std::numbers::pi;

  // Find a direction that is not the normal based off of whether or not the
  // normal's components are all equal to sqrt(1/3) or whether or not at least
  // one component is less than sqrt(1/3). Learned this trick from Peter Kutz.
  glm::vec3 direction_not_normal;
  if (std::abs(normal.x) < SQRT_ONE_THIRD) {
    direction_not_normal = glm::vec3(1, 0, 0);
  } else if (std::abs(normal.y) < SQRT_ONE_THIRD) {
    direction_not_normal = glm::vec3(0, 1, 0);
  } else {
    direction_not_normal = glm::vec3(0, 0, 1);
  }

  // Use not-normal direction to generate two perpendicular directions
  glm::vec3 perp_dir_1 = glm::normalize(glm::cross(normal, direction_not_normal));
  glm::vec3 perp_dir_2 = glm::normalize(glm::cross(normal, perp_dir_1));

  return up * normal + std::cos(around) * over * perp_dir_1 + std::sin(around) * over * perp_dir_2;
}

__host__ __device__ inline void sample_material(int index,
                                                int curr_iter,
                                                int curr_depth,
                                                Material material,
                                                Hit hit,
                                                PathSegment* segments) {
  PathSegment og_segment = segments[index];

  cuda::std::visit(
      Match{
          [=](UnknownMat) {
            segments[index].radiance = 1.f;
            segments[index].throughput = glm::vec3(1.f, 0.f, 1.f);
          },

          [=](Light light) {
            segments[index].radiance = light.emission;
            segments[index].throughput *= light.color;
          },

          [=](Diffuse diffuse) {
            Ray og_ray = og_segment.ray;
            glm::vec3 omega_o = -og_ray.direction;

            // Calculate Lambertian term, which is also is cos(theta)
            float lambert = glm::abs(glm::dot(hit.normal, omega_o));

            // BSDF for perfectly diffuse materials is given by (albedo / pi)
            glm::vec3 bsdf = diffuse.color * static_cast<float>(std::numbers::inv_pi);

            // PDF for cosine-weighted hemisphere sampling
            float pdf = lambert * std::numbers::inv_pi;

            segments[index].throughput *= bsdf * lambert / pdf;

            auto rng = make_seeded_random_engine(curr_iter, index, curr_depth);

            // Determine next ray
            segments[index].ray = {
                .origin = og_ray.get_point(hit.t),
                .direction = calculate_random_direction_in_hemisphere(hit.normal, rng),
            };
          },

          [=](PureReflection specular) {
            Ray og_ray = og_segment.ray;

            segments[index].throughput *= specular.color;
            segments[index].ray = {
                .origin = og_ray.get_point(hit.t),
                .direction = glm::normalize(glm::reflect(og_ray.direction, hit.normal)),
            };
          },

          [=](PureTransmission transmissive) {
            Ray og_ray = og_segment.ray;

            // GLSL/GLM refract expects the IOR ratio to be incident over target, so
            // we treat the default as us starting from inside the material
            float eta = transmissive.eta;
            if (hit.surface == Surface::Outside) {
              eta = 1.f / eta;
            }

            glm::vec3 result = glm::refract(og_ray.direction, hit.normal, eta);

            // Handle total internal reflection
            if (result == glm::vec3()) {
              return;
            }

            segments[index].throughput *= transmissive.color;
            segments[index].ray = {
                // Need to offset origin by an additional factor. Otherwise, it appears that
                // the new origin isn't fully inside the material yet.
                .origin = og_ray.get_point(hit.t) + 0.0001f * og_ray.direction,
                .direction = glm::normalize(result),
            };
          },
      },
      material);
}
