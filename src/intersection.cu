#include "hip/hip_runtime.h"
#include "intersection.hpp"

#include <cuda/std/cmath>
#include <cuda/std/limits>

#include <glm/gtx/intersect.hpp>

__device__ Intersection test_cube_isect(const Geometry& cube, Ray ray) {
  float t_min = cuda::std::numeric_limits<float>::lowest();
  float t_max = cuda::std::numeric_limits<float>::max();
  glm::vec3 t_min_n;
  glm::vec3 t_max_n;

  Ray obj_ray = {
      .origin = glm::vec3(cube.inv_transform * glm::vec4(ray.origin, 1.f)),
      .direction = glm::vec3(cube.inv_transform * glm::vec4(ray.direction, 0.f)),
  };

  for (int xyz = 0; xyz < 3; ++xyz) {
    float qdxyz = obj_ray.direction[xyz];

    /*if (glm::abs(qdxyz) > 0.00001f)*/
    {
      float t1 = (-0.5f - obj_ray.origin[xyz]) / qdxyz;
      float t2 = (0.5f - obj_ray.origin[xyz]) / qdxyz;
      float ta = glm::min(t1, t2);
      float tb = glm::max(t1, t2);

      glm::vec3 n;
      n[xyz] = t2 < t1 ? 1.f : -1.f;

      if (ta > 0.f && ta > t_min) {
        t_min = ta;
        t_min_n = n;
      }

      if (tb < t_max) {
        t_max = tb;
        t_max_n = n;
      }
    }
  }

  Intersection isect;

  if (t_max >= t_min && t_max > 0.f) {
    isect.surface = Surface::Outside;

    if (t_min <= 0.f) {
      t_min = t_max;
      t_min_n = t_max_n;
      isect.surface = Surface::Inside;
    }

    isect.point = glm::vec3(cube.transform * glm::vec4(obj_ray.at(t_min), 1.f));
    isect.normal = glm::normalize(glm::vec3(cube.inv_transpose * glm::vec4(t_min_n, 0.f)));
    isect.t = glm::length(ray.origin - isect.point);
    isect.material_id = cube.material_id;
  } else {
    isect.t = -1.f;
  }

  return isect;
}

__device__ Intersection test_sphere_isect(const Geometry& sphere, Ray ray) {
  static const float radius = 0.5f;

  Intersection isect;
  isect.t = -1.f;

  Ray obj_ray = {
      .origin = glm::vec3(sphere.inv_transform * glm::vec4(ray.origin, 1.f)),
      .direction = glm::normalize(glm::vec3(sphere.inv_transform * glm::vec4(ray.direction, 0.f))),
  };

  float vector_dot_direction = glm::dot(obj_ray.origin, obj_ray.direction);
  float radicand = vector_dot_direction * vector_dot_direction -
                   (glm::dot(obj_ray.origin, obj_ray.origin) - (radius * radius));

  if (radicand < 0.f) {
    return isect;
  }

  float square_root = std::sqrt(radicand);
  float first_term = -vector_dot_direction;
  float t1 = first_term + square_root;
  float t2 = first_term - square_root;

  float t = 0.f;
  if (t1 < 0.f && t2 < 0.f) {
    return isect;
  } else if (t1 > 0.f && t2 > 0.f) {
    t = glm::min(t1, t2);
    isect.surface = Surface::Outside;
  } else {
    // Not sure if this takes into account intersections w.r.t. the tangent
    // of the sphere. Can't just assume we're inside the sphere?
    t = glm::max(t1, t2);
    isect.surface = Surface::Inside;
  }

  glm::vec3 obj_point = obj_ray.at(t);

  isect.point = glm::vec3(sphere.transform * glm::vec4(obj_point, 1.f));
  isect.t = glm::length(ray.origin - isect.point);
  isect.normal = glm::normalize(glm::vec3(sphere.inv_transpose * glm::vec4(obj_point, 0.f)));
  isect.material_id = sphere.material_id;

  if (isect.surface == Surface::Inside) {
    isect.normal = -isect.normal;
  }

  return isect;
}

__device__ Intersection test_tri_list_isect(int tri_idx_begin,
                                            int tri_idx_end,
                                            Ray obj_ray,
                                            const Triangle* triangle_list,
                                            const glm::vec3* position_list,
                                            const glm::vec3* normal_list) {
  Intersection isect;
  isect.t = -1.f;
  float t_min = cuda::std::numeric_limits<float>::max();

  for (int tri_idx = tri_idx_begin; tri_idx < tri_idx_end; ++tri_idx) {
    const Triangle& triangle = triangle_list[tri_idx];
    const glm::vec3 v0 = position_list[triangle[0].pos_idx];
    const glm::vec3 v1 = position_list[triangle[1].pos_idx];
    const glm::vec3 v2 = position_list[triangle[2].pos_idx];

    glm::vec3 bary;
    if (!glm::intersectRayTriangle(obj_ray.origin, obj_ray.direction, v0, v1, v2, bary)) {
      continue;
    }

    if (t_min > bary.z) {
      float u = bary.x;
      float v = bary.y;
      float w = 1.f - u - v;

      const glm::vec3 normal = normal_list[triangle[1].nor_idx];
      const glm::vec3 point = w * v0 + u * v1 + v * v2;

      if (glm::dot(normal, obj_ray.direction) < 0.f) {
        isect.surface = Surface::Outside;
      } else {
        isect.surface = Surface::Inside;
      }

      // Keep in local space, perform transform outside
      isect.normal = normal;
      isect.point = point;
      isect.t = bary.z;

      t_min = isect.t;
    }
  }

  return isect;
}

__device__ Intersection test_gltf_isect(const Geometry& gltf,
                                        Ray ray,
                                        Triangle* triangle_list,
                                        glm::vec3* position_list,
                                        glm::vec3* normal_list) {
  Ray obj_ray = {
      .origin = glm::vec3(gltf.inv_transform * glm::vec4(ray.origin, 1.f)),
      .direction = glm::vec3(gltf.inv_transform * glm::vec4(ray.direction, 0.f)),
  };

  Intersection isect = test_tri_list_isect(gltf.tri_begin, gltf.tri_end, obj_ray, triangle_list,
                                           position_list, normal_list);

  if (isect.t < 0.f) return isect;

  // Transform back to world space
  isect.normal = glm::normalize(glm::vec3(gltf.inv_transpose * glm::vec4(isect.normal, 0.f)));
  isect.point = glm::vec3(gltf.transform * glm::vec4(isect.point, 1.f));
  isect.material_id = gltf.material_id;

  return isect;
}

__device__ Intersection test_bvh_isect(int root_node_idx,
                                       Ray world_ray,
                                       const Geometry& geometry,
                                       const bvh::Node* node_list,
                                       const Triangle* triangle_list,
                                       const glm::vec3* position_list,
                                       const glm::vec3* normal_list) {
  Intersection isect;
  isect.t = -1.f;
  float t_min = cuda::std::numeric_limits<float>::max();

  int to_check[10];
  int idx = 0;
  to_check[idx++] = root_node_idx;

  glm::vec3 inv_direction = 1.f / world_ray.direction;
  Ray obj_ray = {
      .origin = glm::vec3(geometry.inv_transform * glm::vec4(world_ray.origin, 1.f)),
      .direction = glm::vec3(geometry.inv_transform * glm::vec4(world_ray.direction, 0.f)),
  };

  while (idx > 0) {
    const bvh::Node& node = node_list[to_check[--idx]];

    // Hit leaf
    if (node.child_idx == -1) {
      Intersection curr = test_tri_list_isect(node.tri_idx, node.tri_idx + node.tri_count, obj_ray,
                                              triangle_list, position_list, normal_list);

      if (curr.t > 0.f && curr.t < t_min) {
        isect = std::move(curr);
        t_min = isect.t;
      }
    } else {
      const bvh::Node& c0 = node_list[node.child_idx];
      const bvh::Node& c1 = node_list[node.child_idx + 1];

      float t_result_c0 = c0.bbox.intersect(world_ray, inv_direction);
      float t_result_c1 = c1.bbox.intersect(world_ray, inv_direction);

      // Two things: first, we want to always look at the closer child BVH node first, because
      // it may mean we get to skip checking farther BVH nodes later. Secondly, we only
      // check the child nodes if it's closer than our current intersection.
      bool is_c0_nearer = t_result_c0 < t_result_c1;
      float near = is_c0_nearer ? t_result_c0 : t_result_c1;
      float far = is_c0_nearer ? t_result_c1 : t_result_c0;
      int child_idx_near = is_c0_nearer ? node.child_idx : node.child_idx + 1;
      int child_idx_far = is_c0_nearer ? node.child_idx + 1 : node.child_idx;

      if (far < t_min) to_check[idx++] = child_idx_far;
      if (near < t_min) to_check[idx++] = child_idx_near;
    }
  }

  if (isect.t > 0.f) {
    isect.normal = glm::normalize(glm::vec3(geometry.inv_transpose * glm::vec4(isect.normal, 0.f)));
    isect.point = glm::vec3(geometry.transform * glm::vec4(isect.point, 1.f));
    isect.material_id = geometry.material_id;
  }

  return isect;
}

namespace kernel {

__global__ void find_intersections(int num_paths,
                                   Geometry* geometry_list,
                                   int geometry_list_size,
                                   Material* material_list,
                                   Triangle* triangle_list,
                                   glm::vec3* position_list,
                                   glm::vec3* normal_list,
                                   bvh::Node* bvh_node_list,
                                   Triangle* bvh_tri_list,
                                   PathSegment* segments,
                                   Intersection* intersections,
                                   bool bbox_isect_culling,
                                   bool bvh_isect_culling) {
  int segment_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (segment_index >= num_paths) {
    return;
  }

  PathSegment segment = segments[segment_index];

  if (segment.remaining_bounces == 0) {
    return;
  }

  Ray segment_ray = segment.ray;
  glm::vec3 inv_direction = 1.f / segment_ray.direction;
  float t_min = cuda::std::numeric_limits<float>::max();

  Intersection isect;
  isect.t = -1.f;

  for (int geometry_index = 0; geometry_index < geometry_list_size; ++geometry_index) {
    const Geometry& geometry = geometry_list[geometry_index];

    if (bbox_isect_culling) {
      float t_result = geometry.bbox.intersect(segment_ray, inv_direction);

      if (cuda::std::isinf(t_result) || t_result >= t_min) continue;
    }

    Intersection curr_isect;

    switch (geometry.type) {
      case Geometry::Type::Cube:
        curr_isect = test_cube_isect(geometry, segment_ray);
        break;

      case Geometry::Type::Sphere:
        curr_isect = test_sphere_isect(geometry, segment_ray);
        break;

      case Geometry::Type::Gltf: {
        if (bvh_isect_culling && geometry.bvh_root_idx >= 0) {
          curr_isect = test_bvh_isect(geometry.bvh_root_idx, segment_ray, geometry, bvh_node_list,
                                      bvh_tri_list, position_list, normal_list);
        } else {
          curr_isect =
              test_gltf_isect(geometry, segment_ray, triangle_list, position_list, normal_list);
        }

        break;
      }

      default:
        // Unreachable
        return;
    }

    // Ray did not hit any geometry
    if (curr_isect.t < 0.f) {
      continue;
    }

    // Discovered a closer object, save it
    if (t_min > curr_isect.t) {
      t_min = curr_isect.t;
      isect = curr_isect;
    }
  }

  intersections[segment_index] = std::move(isect);
}

}  // namespace kernel
